#include "hip/hip_runtime.h"
//
// Created by ashish on 4/4/17.
//

//TODO remove this header
#include ""


#include "BasicScene.hpp"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "math.h"
#include <math_functions.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>
#include "vector_functions.h"
#include ""
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "cudaUtils.h"





  // Size of the traversal stack in local memory.
//texture<float4, 1, hipReadModeElementType> bvhNodesTexture;
//texture<float4, 1, hipReadModeElementType> triWoopTexture;
//texture<float4, 1, hipReadModeElementType> triNormalsTexture;
//texture<int, 1, hipReadModeElementType> triIndicesTexture;
//
//__device__ int counter = 0;
//>>>>>>> accelerated_BVH


__global__ void cudaProcess(const kernelInfo info){




    uint tx = threadIdx.x;
    uint ty = threadIdx.y;
    uint bw = blockDim.x;
    uint bh = blockDim.y;
    uint x = blockIdx.x*bw + tx;
    uint y = blockIdx.y*bh + ty;
    size_t pixelPos = y*info.width+x;
    const glm::vec4 * const triTex = info.triangleTex;
    const Sphere * const sphereTex = info.sphereTex;

    const size_t triTexSize = info.numVerts;
    const size_t sphTexSize = info.numSpheres;

    const int depth = info.depth;
    const int w = info.width;
    const int h = info.height;




    hiprandState randState; // state of the random number generator, to prevent repetition
    hiprand_init(info.hash + (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x, 0, 0, &randState);

    if(x == 0 && y ==0 ) {

//        printf("rei tex size is %ld \n",info.numVerts);
        //TODO keep this function and disable it
//        printf("received vars\n");
//        printf("%f\n",info.cam.dist);
//        printf("%f\n",info.cam.fov);
//        printf("%f\n",info.cam.aspect);
//        printf("%d\n",info.width);
//        printf("%d\n",info.height);
//        printf("cam width %f\n",info.cam.dist*info.cam.aspect*info.cam.fov);
//        printf("tri tex size %ld\n",info.numVerts);
    }

    if(x>=w || y>=h)
        return;


    //test texture cuda

//    float4 col  = tex1Dfetch<float4>(info.triangleTex,pixelPos);
//    glm::vec4 newVec = f4tov(col);
//    col = vtof4(newVec);


    u_char r = 255,g = 255,b = 255,a = 255;

    Ray camRay = getCamRayDir(info.cam,x,y,w,h);
    
    
    
    {
        vec3 mask = vec3(1.0f, 1.0f, 1.0f); // colour mask
        vec3 accucolor = vec3(0.0f, 0.0f, 0.0f); // accumulated colour
        vec3 direct = vec3(0, 0, 0);

        for (unsigned int d = 0; d < depth; ++d){

            int minSphereIdx = -1;
            int minBoxId = -1;
            int minTriIdx = -1;
            int geomtype = GeoType::NONE;
            float scene_t = 1e20;
            vec3 objcol = vec3(0, 0, 0);
            vec3 emit = vec3(0, 0, 0);
            vec3 hitpos;//pos in 3d where ray hit the closest

            vec3 n; // normal
            vec3 nl; // oriented normal
            vec3 nextdir; // ray direction of next path segment
            vec3 trinormal = vec3(0, 0, 0);
            Mat mat;
            //TODO this magic num
            float tmin = 0.00001f; // set to 0.01f when using refractive material
            float tmax = 1e20;
            //TODO remove this debug
            int  debug = 0;

//            intersectAllTriangles(triTex,camRay,scene_t,minTriIdx,triTexSize,geomtype,info.cullBackFaces);
            intersectBVHandTriangles(glm::vec4(camRay.origin,0),glm::vec4(camRay.dir,0),info.bvhData.dev_triNode,info.bvhData.dev_triWoopTpr, nullptr,info.bvhData.dev_triIndicesTpr,minTriIdx,scene_t,debug,n,info.bvhData.leafCount,info.bvhData.triCount,false);


            if(minTriIdx==-1){
                scene_t = min(scene_t,45.0f);
                scene_t = (scene_t-15)/30;
                r = 255*scene_t;
                g=0;

            }
//            intersectAllSpeheres(sphereTex,camRay,scene_t,minSphereIdx,sphTexSize,geomtype);

//            if(geomtype == GeoType::SPHERE){
//                const Sphere & hS = sphereTex[minSphereIdx];
//                hitpos = camRay.origin+camRay.dir*scene_t;
//                vec3 n = hS.getNormal(hitpos);
//
//                //TODO see this inversion later for culling
//                vec3 nl = glm::dot(n, camRay.dir) < 0 ? n : n * -1.0f;
//                objcol = vec3(hS.col.x, hS.col.y,hS.col.z);   // object colour
//                emit = vec3(hS.emi.x, hS.emi.y, hS.emi.z);  // object emission
//                mat = hS.mat;
//                accucolor += (mask * emit);
//
//            }
//            else if(geomtype == GeoType::TRI){
//                r = 128;
//                pBestTri = &triTex[hitTriIdx];
//                hitpoint = rayorig + raydir * scene_t; // intersection point
//
//                // float4 normal = tex1Dfetch(triNormalsTexture, pBestTriIdx);
//                n = trinormal;
//                n.normalize();
//                nl = dot(n, raydir) < 0 ? n : n * -1;  // correctly oriented normal
//                //vec3 colour = hitTriIdx->_colorf;
//                vec3 colour = vec3(0.9f, 0.3f, 0.0f); // hardcoded triangle colour  .9f, 0.3f, 0.0f
//                refltype = COAT; // objectmaterial
//                objcol = colour;
//                emit = vec3(0.0, 0.0, 0);  // object emission
//                accucolor += (mask * emit);

//            }
//            else if(geomtype == GeoType::BOX){
//                Box &box = boxes[box_id];
//                x = r.orig + r.dir*t;  // intersection point on object
//                n = normalize(box.normalAt(x)); // normal
//                nl = dot(n, r.dir) < 0 ? n : n * -1;  // correctly oriented normal
//                f = box.col;  // box colour
//                refltype = box.refl;
//                emit = box.emi; // box emission
//                accucolor += (mask * emit);


//            }



            if (mat == DIFF){

//                // pick two random numbers
//                float phi = 2 * M_PI * hiprand_uniform(randstate);
//                float r2 = hiprand_uniform(randstate);
//                float r2s = sqrtf(r2);
//
//                // compute orthonormal coordinate frame uvw with hitpoint as origin
//                vec3 w = nl; w.normalize();
//                vec3 u = cross((fabs(w.x) > .1 ? vec3(0, 1, 0) : vec3(1, 0, 0)), w); u.normalize();
//                vec3 v = cross(w, u);
//
//                // compute cosine weighted random ray direction on hemisphere
//                nextdir = u*cosf(phi)*r2s + v*sinf(phi)*r2s + w*sqrtf(1 - r2);
//                nextdir.normalize();
//
//                // offset origin next path segment to prevent self intersection
//                hitpoint += nl * 0.001f; // scene size dependent
//
//                // multiply mask with colour of object
//                mask *= objcol;

            } // end diffuse material
//
//            // Phong metal material from "Realistic Ray Tracing", P. Shirley
//            if (refltype == METAL){
//
//                // compute random perturbation of ideal reflection vector
//                // the higher the phong exponent, the closer the perturbed vector is to the ideal reflection direction
//                float phi = 2 * M_PI * hiprand_uniform(randstate);
//                float r2 = hiprand_uniform(randstate);
//                float phongexponent = 30;
//                float cosTheta = powf(1 - r2, 1.0f / (phongexponent + 1));
//                float sinTheta = sqrtf(1 - cosTheta * cosTheta);
//
//                // create orthonormal basis uvw around reflection vector with hitpoint as origin
//                // w is ray direction for ideal reflection
//                vec3 w = raydir - n * 2.0f * dot(n, raydir); w.normalize();
//                vec3 u = cross((fabs(w.x) > .1 ? vec3(0, 1, 0) : vec3(1, 0, 0)), w); u.normalize();
//                vec3 v = cross(w, u); // v is already normalised because w and u are normalised
//
//                // compute cosine weighted random ray direction on hemisphere
//                nextdir = u * cosf(phi) * sinTheta + v * sinf(phi) * sinTheta + w * cosTheta;
//                nextdir.normalize();
//
//                // offset origin next path segment to prevent self intersection
//                hitpoint += nl * 0.0001f;  // scene size dependent
//
//                // multiply mask with colour of object
//                mask *= objcol;
//            }
//
//            // ideal specular reflection (mirror)
//            if (refltype == SPEC){
//
//                // compute relfected ray direction according to Snell's law
//                nextdir = raydir - n * dot(n, raydir) * 2.0f;
//                nextdir.normalize();
//
//                // offset origin next path segment to prevent self intersection
//                hitpoint += nl * 0.001f;
//
//                // multiply mask with colour of object
//                mask *= objcol;
//            }


            // COAT material based on https://github.com/peterkutz/GPUPathTracer
            // randomly select diffuse or specular reflection
            // looks okay-ish but inaccurate (no Fresnel calculation yet)
//            if (refltype == COAT){
//
//                float rouletteRandomFloat = hiprand_uniform(randstate);
//                float threshold = 0.05f;
//                vec3 specularColor = vec3(1, 1, 1);  // hard-coded
//                bool reflectFromSurface = (rouletteRandomFloat < threshold); //computeFresnel(make_vec3(n.x, n.y, n.z), incident, incidentIOR, transmittedIOR, reflectionDirection, transmissionDirection).reflectionCoefficient);
//
//                if (reflectFromSurface) { // calculate perfectly specular reflection
//
//                    // Ray reflected from the surface. Trace a ray in the reflection direction.
//                    // TODO: Use Russian roulette instead of simple multipliers!
//                    // (Selecting between diffuse sample and no sample (absorption) in this case.)
//
//                    mask *= specularColor;
//                    nextdir = raydir - n * 2.0f * dot(n, raydir);
//                    nextdir.normalize();
//
//                    // offset origin next path segment to prevent self intersection
//                    hitpoint += nl * 0.001f; // scene size dependent
//                }
//
//                else {  // calculate perfectly diffuse reflection
//
//                    float r1 = 2 * M_PI * hiprand_uniform(randstate);
//                    float r2 = hiprand_uniform(randstate);
//                    float r2s = sqrtf(r2);
//
//                    // compute orthonormal coordinate frame uvw with hitpoint as origin
//                    vec3 w = nl; w.normalize();
//                    vec3 u = cross((fabs(w.x) > .1 ? vec3(0, 1, 0) : vec3(1, 0, 0)), w); u.normalize();
//                    vec3 v = cross(w, u);
//
//                    // compute cosine weighted random ray direction on hemisphere
//                    nextdir = u*cosf(r1)*r2s + v*sinf(r1)*r2s + w*sqrtf(1 - r2);
//                    nextdir.normalize();
//
//                    // offset origin next path segment to prevent self intersection
//                    hitpoint += nl * 0.001f;  // // scene size dependent
//
//                    // multiply mask with colour of object
//                    mask *= objcol;
//                }
//            } // end COAT

            // perfectly refractive material (glass, water)
            // set ray_tmin to 0.01 when using refractive material
//            if (refltype == REFR){
//
//                bool into = dot(n, nl) > 0; // is ray entering or leaving refractive material?
//                float nc = 1.0f;  // Index of Refraction air
//                float nt = 1.4f;  // Index of Refraction glass/water
//                float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
//                float ddn = dot(raydir, nl);
//                float cos2t = 1.0f - nnt*nnt * (1.f - ddn*ddn);
//
//                if (cos2t < 0.0f) // total internal reflection
//                {
//                    nextdir = raydir - n * 2.0f * dot(n, raydir);
//                    nextdir.normalize();
//
//                    // offset origin next path segment to prevent self intersection
//                    hitpoint += nl * 0.001f; // scene size dependent
//                }
//                else // cos2t > 0
//                {
//                    // compute direction of transmission ray
//                    vec3 tdir = raydir * nnt;
//                    tdir -= n * ((into ? 1 : -1) * (ddn*nnt + sqrtf(cos2t)));
//                    tdir.normalize();
//
//                    float R0 = (nt - nc)*(nt - nc) / (nt + nc)*(nt + nc);
//                    float c = 1.f - (into ? -ddn : dot(tdir, n));
//                    float Re = R0 + (1.f - R0) * c * c * c * c * c;
//                    float Tr = 1 - Re; // Transmission
//                    float P = .25f + .5f * Re;
//                    float RP = Re / P;
//                    float TP = Tr / (1.f - P);
//
//                    // randomly choose reflection or transmission ray
//                    if (hiprand_uniform(randstate) < 0.2) // reflection ray
//                    {
//                        mask *= RP;
//                        nextdir = raydir - n * 2.0f * dot(n, raydir);
//                        nextdir.normalize();
//
//                        hitpoint += nl * 0.001f; // scene size dependent
//                    }
//                    else // transmission ray
//                    {
//                        mask *= TP;
//                        nextdir = tdir;
//                        nextdir.normalize();
//
//                        hitpoint += nl * 0.001f; // epsilon must be small to avoid artefacts
//                    }
//                }
//            }
//
//            // set up origin and direction of next path segment
//            rayorig = hitpoint;
//            raydir = nextdir;
        }


    }

    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
    
//    {
//        float t;
//        int triangle_id;
//        int geomtype = -1;
//
//        float tmin = 1e20;
//        float tmax = -1e20;
//
//        float d = 1e21;
//        float k = 1e21;
//        float q = 1e21;
//        float inf = t = 1e20;
//
//
//
//
//
//        // if ray hits bounding box of triangle meshes, intersect ray with all triangles
//        //TODO insert bounding box here
//        intersectAllTriangles(triTex,camRay, t, triangle_id, triTexSize, geomtype,info.cullBackFaces);
//
//
//
//
//
//
//        if(t<inf){
//            t = min(45.0f,t);
//            t-=15;
//            t/=30;
//            r = 255*t;
//            g = 0;
//            b = 0;
//            a = 255;
//        }



//         t is distance to closest intersection of ray with all primitives in the scene (spheres, boxes and triangles)
//        return t<inf;

//    }



    //sphere test
//    {
//
//        float rad= 300/(sqrt(2.0f)-1);
//        Sphere sp(rad/2,vec3(0.0f, 0,-rad-h/2),vec3(0,0,0),vec3(0.9f, 0.9f, 0.9f ), DIFF);
//        float dist = sp.intersect(camRay);
//
//        if(dist > 0 ){
//            r = 0;
//            g = 255;
//            b = 0;
//            a = 255;
//        }
//    }


    uchar4 c4 = make_uchar4(r, g, b, a);
    info.dev_drawRes[pixelPos] = rgbToUint(c4.x,c4.y,c4.z);

}

#include <iostream>
void BasicScene::launchKernel(const kernelInfo &info) {

//    using namespace std;
//    cout << width <<" " << height << endl;


     dim3 blocks((info.width+info.blockSize.x)/info.blockSize.x,(info.height+info.blockSize.y)/info.blockSize.y,1);
     cudaProcess<<<blocks,info.blockSize>>>(info);
}


